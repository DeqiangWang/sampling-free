#include "hip/hip_runtime.h"
// Joya Chen
// chenjoya@mail.ustc.edu.cn / chenjoya@foxmail.com
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

#include <cfloat>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__global__ void AloneBCELossForward(const int nthreads, 
    const T* logits,
    const int* targets,
    const int num_classes,
    const int num, 
    T* losses) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {

    int n = i / num_classes;
    int d = i % num_classes; // current class[0~79]; 
    int t = targets[n]; // target class [1~80];

    // Decide it is positive or negative case. 
    T c1 = (t == (d+1)); 
    T c2 = (t>=0 & t != (d+1));

    T  p = 1. / (1. + expf(-logits[i]));

    T term1 = logf(max(p, FLT_MIN));

    T term2 = -1. * logits[i] * (logits[i] >= 0) - logf(1. + expf(logits[i] - 2. * logits[i] * (logits[i] >= 0)));

    losses[i] = 0.0;
    losses[i] += -c1 * term1;
    losses[i] += -c2 * term2;

  } // CUDA_1D_KERNEL_LOOP
} // AloneBCELossForward


template <typename T>
__global__ void AloneBCELossBackward(const int nthreads,
                const T* logits,
                const int* targets,
                const T* d_losses,
                const int num_classes,
                const int num,
                T* d_logits) {
  CUDA_1D_KERNEL_LOOP(i, nthreads) {

    int n = i / num_classes;
    int d = i % num_classes; // current class[0~79]; 
    int t = targets[n]; // target class [1~80], 0 is background;

    // Decide it is positive or negative case. 
    T c1 = (t == (d+1));
    T c2 = (t>=0 & t != (d+1));

    T  p = 1. / (1. + expf(-logits[i]));

    T term1 = 1. - p;

    T term2 = 0. - p;

    d_logits[i] = 0.0;
    d_logits[i] += -c1 * term1;
    d_logits[i] += -c2 * term2;
    d_logits[i] *= d_losses[i];

  } // CUDA_1D_KERNEL_LOOP
} // AloneBCELossBackward


at::Tensor AloneBCELoss_forward_cuda(
		const at::Tensor& logits,
                const at::Tensor& targets,
		const int num_classes) {
  AT_ASSERTM(logits.type().is_cuda(), "logits must be a CUDA tensor");
  AT_ASSERTM(targets.type().is_cuda(), "targets must be a CUDA tensor");
  AT_ASSERTM(logits.dim() == 2, "logits should be NxClass");

  const int num_samples = logits.size(0);
	
  auto losses = at::empty({num_samples, logits.size(1)}, logits.options());
  auto losses_size = num_samples * logits.size(1);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)losses_size, 512L), 4096L));
  
  dim3 block(512);

  if (losses.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return losses;
  }

  AT_DISPATCH_FLOATING_TYPES(logits.type(), "AloneBCELoss_forward", [&] {
    AloneBCELossForward<scalar_t><<<grid, block, 0, stream>>>(
         losses_size,
         logits.contiguous().data<scalar_t>(),
	 targets.contiguous().data<int>(),
         num_classes,
	 num_samples,
         losses.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return losses;   
}	


at::Tensor AloneBCELoss_backward_cuda(
		const at::Tensor& logits,
                const at::Tensor& targets,
		const at::Tensor& d_losses,
		const int num_classes) {
  AT_ASSERTM(logits.type().is_cuda(), "logits must be a CUDA tensor");
  AT_ASSERTM(targets.type().is_cuda(), "targets must be a CUDA tensor");
  AT_ASSERTM(d_losses.type().is_cuda(), "d_losses must be a CUDA tensor");

  AT_ASSERTM(logits.dim() == 2, "logits should be NxClass");

  const int num_samples = logits.size(0);
  AT_ASSERTM(logits.size(1) == num_classes, "logits.size(1) should be num_classes");
	
  auto d_logits = at::zeros({num_samples, num_classes}, logits.options());
  auto d_logits_size = num_samples * logits.size(1);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv((long)d_logits_size, 512L), 4096L));
  dim3 block(512);

  if (d_logits.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return d_logits;
  }

  AT_DISPATCH_FLOATING_TYPES(logits.type(), "AloneBCELoss_backward", [&] {
    AloneBCELossBackward<scalar_t><<<grid, block, 0, stream>>>(
         d_logits_size,
         logits.contiguous().data<scalar_t>(),
	 targets.contiguous().data<int>(),
	 d_losses.contiguous().data<scalar_t>(),
         num_classes,
	 num_samples,
         d_logits.data<scalar_t>());
  });

  THCudaCheck(hipGetLastError());
  return d_logits;   
}	

